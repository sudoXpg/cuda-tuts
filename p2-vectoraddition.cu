/*
    Compile instructions:
    nvcc -g  p2-vectoraddition.cu -o p2-vectoraddition

    To measure performance:
    nvprof ./p2-vectoraddition
*/



#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 1000000

void vector_add_cpu(float *res, float *a, float *b, int n){
    for(int i=0;i<n;i++){
        res[i] = a[i] + b[i];
    }
}



__global__ void vector_add_gpu(float *res, float *a, float *b, int n){
    for(int i=0;i<n;i++){
        res[i] = a[i] + b[i];
    }
}


int main(void){
    float *h_a, *h_b, *h_res;
    float *d_a, *d_b, *d_res;


    // host CPU malloc
    h_a = (float *)malloc(  sizeof(float) * N);
    h_b = (float *)malloc(  sizeof(float) * N);
    h_res = (float *)malloc(sizeof(float) * N);

    for (int i=0;i<N;i++){
        h_a[i] = (float)rand() / (float)RAND_MAX;;
        h_b[i] = (float)rand() / (float)RAND_MAX;;
    }


    // device GPU malloc
    hipMalloc((void **)&d_a, (sizeof(float) * N) );
    hipMalloc((void **)&d_b, (sizeof(float) * N) );
    hipMalloc((void **)&d_res, (sizeof(float) * N) );


    // copy to device
    hipMemcpy(d_a, h_a, N, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, N, hipMemcpyHostToDevice);
    hipMemcpy(d_res, h_res, N, hipMemcpyHostToDevice);



    vector_add_gpu<<<1,1>>>(d_res, d_a, d_b, N);
    hipDeviceSynchronize();

    //vector_add_cpu(h_res, h_a, h_b, N);


    free(h_a);
    free(h_b);
    free(h_res);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_res);

    return 0;
}